#include "hip/hip_runtime.h"
#include "vector_t.h"

__host__ __device__ vector_t::vector_t(void)
{
}

__host__ __device__ vector_t::vector_t(vertex_t origin, vertex_t destination){
	x0 = 0;
	y0 = 0;
	z0 = 0;
	xt = destination.x - origin.x;
	yt = destination.y - origin.y;
	zt = destination.z - origin.z;
}

__host__ __device__ vector_t::vector_t(float x0, float y0, float z0, float xt, float yt, float zt)
{
	this->x0 = x0;
	this->y0 = y0;
	this->z0 = z0;
	this->xt = xt;
	this->yt = yt;
	this->zt = zt;
}

__host__ __device__ float vector_t::calculateDistance(float t){
	float xdist = xt*t;
	float ydist = yt*t;
	float zdist = zt*t;
	return sqrt(xdist*xdist+ydist*ydist+zdist*zdist);
}

__host__ __device__ vertex_t vector_t::getPosAtParameter(float t){
	vertex_t v;
	v.x = x0 + xt*t;
	v.y = y0 + yt*t;
	v.z = z0 + zt*t;
	return v;
}

__host__ __device__ float vector_t::directionDotProduct(vector_t dotterand){
	return xt*dotterand.xt + yt*dotterand.yt + zt*dotterand.zt;
}

__host__ __device__ vector_t vector_t::directionCrossProduct(vector_t crosserand){
	vector_t vector;
	vector.x0 = 0;
	vector.y0 = 0;
	vector.z0 = 0;
	vector.xt = yt*crosserand.zt - zt*crosserand.yt;
	vector.yt = zt*crosserand.xt - xt*crosserand.zt;
	vector.zt = xt*crosserand.yt - yt*crosserand.xt;
	return vector;
}

__host__ __device__ float vector_t::directionMagnitude(void){
	return sqrt(xt*xt+yt*yt+zt*zt);
}

__host__ __device__ vector_t::~vector_t(void)
{
}
