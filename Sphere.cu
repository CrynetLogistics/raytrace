#include "hip/hip_runtime.h"
#include "Sphere.h"

#define PI 3.14159
#define SPHERICAL_MAP 0

__device__ Sphere::Sphere(float centreX, float centreY, float centreZ, float radius, colour_t colour, materialType_t materialType){
	material.initMaterial(materialType);
	centre.x = centreX;
	centre.y = centreY;
	centre.z = centreZ;
	this->radius = radius;
	this->colour.r = colour.r;
	this->colour.g = colour.g;
	this->colour.b = colour.b;
}

__device__ Sphere::Sphere(float centreX, float centreY, float centreZ, float radius, colour_t colour, uint32_t* textureData){
	material.initMaterial(textureData);
	centre.x = centreX;
	centre.y = centreY;
	centre.z = centreZ;
	this->radius = radius;
	this->colour.r = colour.r;
	this->colour.g = colour.g;
	this->colour.b = colour.b;
}

__device__ float Sphere::getRadius(void){
	return radius;
}

__device__ vertex_t Sphere::getCentre(void){
	return centre;
}

__device__ float Sphere::getIntersectionParameter(vector_t lightRay){
	float acx = lightRay.x0-centre.x;
	float acy = lightRay.y0-centre.y;
	float acz = lightRay.z0-centre.z;
	float a = lightRay.xt*lightRay.xt+lightRay.yt*lightRay.yt+lightRay.zt*lightRay.zt;
	float b = 2*(lightRay.xt*acx + lightRay.yt*acy + lightRay.zt*acz);
	float c = acx*acx + acy*acy + acz*acz - radius*radius;
	
	//std::cout<<sqrt(b*b-4*a*c);
	if(b*b-4*a*c>=0){
		float t = (-b-sqrt(b*b-4*a*c))/(2*a);
		//std::cout<<t/1000;
		//return t*60;
		return t;//lightRay.calculateDistance(t);
	}else{
		return 0;
	}
}

//returns a negative for false and positive for true
//magnitude of the number from 0 to 1 to indicate distance from extremum
__device__ float Sphere::getShadowedStatus(vector_t lightRay, float t, Light light){
	vertex_t pos = lightRay.getPosAtParameter(t);
	vector_t normalVector(pos.x, pos.y, pos.z, pos.x-centre.x, pos.y-centre.y, pos.z-centre.z);
	vector_t lightVector(pos.x, pos.y, pos.z, light.getPos().x-pos.x, light.getPos().y-pos.y, light.getPos().z-pos.z);
	vector_t cameraVector(pos.x, pos.y, pos.z, -1*lightRay.xt, -1*lightRay.yt, -1*lightRay.zt);
	float index = normalVector.directionDotProduct(cameraVector)*normalVector.directionDotProduct(lightVector);
	float extremumProjection = normalVector.directionDotProduct(lightVector)/(normalVector.directionMagnitude()*lightVector.directionMagnitude());
	if(index>0){
		return -1 * abs(extremumProjection);// false;
	}else{
		return abs(extremumProjection);//true;
	}
}

//TODO:CURRENTLY UNUSED FEATURE - NORMAL RETURNED IN THE SAME DIRECTION AS REFLECTED RAY
__device__ vector_t Sphere::getNormal(vertex_t pos, vector_t incoming){
	vector_t normalVector(pos.x, pos.y, pos.z, pos.x-centre.x, pos.y-centre.y, pos.z-centre.z);
	//if(normalVector.directionDotProduct(incoming)>0){
	//	normalVector.xt = -1*normalVector.xt;
	//	normalVector.yt = -1*normalVector.yt;
	//	normalVector.zt = -1*normalVector.zt;
	//}
	return normalVector;
}

__device__ colour_t Sphere::getColour(vertex_t position){
	if(material.isTextured()>=2){
		vector_t r(centre, position);
		vector_t s(centre, position);
		s.zt = 0;

		vector_t k_unit(0,0,0,0,0,1);
		vector_t i_unit(0,0,0,1,0,0);

		float cosine_phi = k_unit.directionDotProduct(r)/r.directionMagnitude();
		float cosine_theta = i_unit.directionDotProduct(s)/s.directionMagnitude();

		int x;
		if(SPHERICAL_MAP){
			if(s.yt>0){//theta < pi
				x = 300*(1-acosf(cosine_theta)/PI);
			}else{//theta >= pi
				x = 300*acosf(cosine_theta)/PI;
			}
		}else{
			x = 600*(cosine_theta+1)/2;
		}
		int y = 300*(cosine_phi+1)/2;

		colour.r = (material.getTexture()[600*y+x] & 0x000000FF) >> 0;
		colour.g = (material.getTexture()[600*y+x] & 0x0000FF00) >> 8;
		colour.b = (material.getTexture()[600*y+x] & 0x00FF0000) >> 16;
		return colour;
	}else{
		return colour;
	}
}

__device__ Material Sphere::getMaterial(void){
	return material;
}

__device__ Sphere::~Sphere(void){
}
